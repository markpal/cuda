#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib> // for random numbers
#include <omp.h>
#include <vector>
#include <cstring> // for strcpy
#include <string>
#include <ctime>     // for time()

#define BLOCK_SIZE 32
int N = 1000;

using namespace std;

// -------------------------------------------------- pairing
int paired(char a1, char a2)
{
  if(a1 == 'A' && a2 == 'U')
    return 1;
  if(a1 == 'U' && a2 == 'A')
    return 1;
  if(a1 == 'G' && a2 == 'C')
    return 1;
  if(a1 == 'C' && a2 == 'G')
    return 1;

  return 0;
}

__device__ int _paired(char a, char b) {
  if ((a == 'A' && b == 'U') || (a == 'U' && b == 'A') || (a == 'C' && b == 'G') || (a == 'G' && b == 'C')) {
    return 1;
  }
  return 0;
}

// --------------------------------------------------
// KERNEL

__global__ void myKernel(int **B, int N, int c0, char* seqq)
{
        int c1 = blockIdx.x + c0;
        int bb = BLOCK_SIZE;
        __shared__ int C[BLOCK_SIZE][BLOCK_SIZE];

        if(c1 <= min((N - 1) / bb, (N + c0 - 2 )/ bb))
        //for (int c1 = c0; c1 <= min((N - 1) / 16, (N + c0 - 2 )/ 16); c1 += 1) // parallel loop  blocks
        {
            int _sj = c1-c0;
            int _si = c1;


         for (int m = _sj+1; m < _si; ++m) {

           // Thread row and column
               int row = threadIdx.y;
               int col = threadIdx.x;

              __shared__ int * A_elements[BLOCK_SIZE];
              __shared__ int * B_elements[BLOCK_SIZE];

              A_elements[row] = &B[BLOCK_SIZE * _sj+row][BLOCK_SIZE * m -1];
              B_elements[row] = &B[BLOCK_SIZE * m +row][BLOCK_SIZE * _si];

             if(row < BLOCK_SIZE && col < BLOCK_SIZE){

              register int Cvalue = 0;

              __syncthreads();

              #pragma unroll
              for (int e = 0; e < BLOCK_SIZE; e++)
              {
                  Cvalue = max(A_elements[row][e] + B_elements[e][col], Cvalue);
              }

              __syncthreads();

                C[row][col] = max(C[row][col], Cvalue);

            }

           }

            for (int c2 = max(1, bb * c0 - bb - 1);
                 c2 <= min(bb * c0 + bb - 1, N + bb * c0 - bb * c1 - 1); c2 += 1) { // serial loop
                if (c0 >= 1) {
                    //    #pragma omp parallel for
                    int lb = max(bb * c1, -bb * c0 + bb * c1 + c2);
                    int ub = min(min(N - 1, bb * c1 + bb-1), -bb * c0 + bb * c1 + c2 + bb-1);
                    int c3 = threadIdx.x+ lb;
                    if(c3<=ub) {

                      register int z = B[-c2 + c3][c3];
                     // for (int c3 = max(16 * c1, -16 * c0 + 16 * c1 + c2); c3 <= min(min(N - 1, 16 * c1 + 15), -16 * c0 + 16 * c1 + c2 + 15); c3 += 1) {   // parallel loop threads

                      // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
                      if(1==1){


                        if(threadIdx.y ==0){

                          int _j = (-c2+c3) % BLOCK_SIZE;
                          int _i = c3 % BLOCK_SIZE;


                          for (int c4 = 0; c4 < bb-1; c4 += 1)  // blocks 0 (triangles)
                            z = max(B[-c2 + c3][-c2 + c3 + c4 ] + B[-c2 + c3 + c4 + 1][c3], z);

                          z = max(z, C[_j][_i]); // middle blocks

                         int fragment = (c1 == N/BLOCK_SIZE-1); // last column

                        for (int c4 =  c2 - bb - fragment; c4 < c2; c4 += 1)   // current tile
                          z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3], z);

                          B[-c2 + c3][c3] = max(z,
                                               B[-c2 + c3 + 1][c3 - 1] +  _paired(seqq[-c2 + c3] , seqq[c3] ));
                          }
                      }

                      else // original generated code
                        {
                        for (int c4 = 0; c4 < c2; c4 += 1) {  // serial
                          z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3],  z);
                        }
                        B[-c2 + c3][c3] = max(z,
                                              B[-c2 + c3 + 1][c3 - 1] + _paired(seqq[-c2 + c3], seqq[c3]));
                        }
                      }

                } else {
                    //  #pragma omp parallel for
                  int lb = bb * c1 + c2;
                  int ub = min(N - 1, bb * c1 + bb-1);
                  int c3 = threadIdx.x + lb;  // threadIdx.x
                  if(c3<=ub) {
                  //for (int c3 = 16 * c1 + c2; c3 <= min(N - 1, 16 * c1 + 15); c3 += 1) {   // parallel loop threads
                    register int z = B[-c2 + c3][c3];
                        for (int c4 = 0; c4 < c2; c4 += 1) {  // serial
                            z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3],  z);
                        }
                        B[-c2 + c3][c3] = max(z,
                                              B[-c2 + c3 + 1][c3 - 1] + _paired(seqq[-c2 + c3], seqq[c3]));

                    }

                }
            }
        }

}


// --------------------------------------------------


int main() {



 // string seq = "UCGCUACCAUUGCUUCUAGACCUACGAAAUAGUCUCAUCUCUACGGCAGUAGUGCAUCUGUGUCGCGCUGUUCGUGAACCGAGACGUUGCAAGUCUUGUGUCAUUUAGGCGUAUGCACUGCUCUCCCU";
   string seq = "GUACGUACGUACGUACGUAC";
  seq = "CUGGUUUAUGUCACCCAGCAGCAGACCCUCCUUUACCGAAAGAUGAUGCUCGUAUUAUUGUACG";
  N += BLOCK_SIZE - N % BLOCK_SIZE;
 //int N = seq.length();


  int n = N, i,j,k;

  char *seqq = new char[N+1];
  if(N>1) // no debug
   {
    char znaki[] = {'C', 'G', 'U', 'A'};
    srand(static_cast<unsigned int>(time(0)));

    for (int i = 0; i < N; i++) {
      seqq[i] = znaki[rand() % 4];  // Losowy wybór z zestawu 'C', 'G', 'U', 'A'
    }
   }
   cout << seqq << endl;
  std::strcpy(seqq, seq.c_str());          // Copy the string content   // use random data for given big N, comment this

  int* flatArray_S = new int[n * n];
  int* flatArray_S_CPU = new int[n * n];

  // Allocate 2D host array for CPU and GPU
  int** S = new int*[n];
  int** S_CPU = new int*[n];

  for(int i = 0; i < n; i++) {
    S[i] = &flatArray_S[i * n];
    S_CPU[i] = &flatArray_S_CPU[i * n];
  }
  // initialization
  for(i=0; i<N; i++) {
    for(j=0; j<N; j++){
      S[i][j] = INT_MIN;
      S_CPU[i][j] = INT_MIN;
    }
  }
  for(i=0; i<N; i++){
    S[i][i] = 0;
    S_CPU[i][i] = 0;
    if(i+1 < N) {
      S[i][i + 1] = 0;
      S[i+1][i] = 0;
      S_CPU[i][i+1] = 0;
      S_CPU[i+1][i] = 0;
    }
  }
  // -----------------------------

  // cuda memory allocation
  int* flat_d_S;
  int** d_S;
  char *d_sequence;

  double start_time = omp_get_wtime();
  hipMalloc(&d_sequence, n);
  hipMalloc(&flat_d_S, n * n * sizeof(int));
  hipMalloc(&d_S, n * sizeof(int*));

  int* h_S[n];  // copy flat_d_S pointers to vector on host and copy to d_S vector of pointers
  for(int i = 0; i < n; i++) {
    h_S[i] = flat_d_S + i * n;
  }
  hipMemcpy(d_S, h_S, n * sizeof(int*), hipMemcpyHostToDevice);
  hipMemcpy(d_sequence, seqq, n, hipMemcpyHostToDevice);
  // Copy host data to device before entering the loop
  hipMemcpy(flat_d_S, &S[0][0], n * n * sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = (n) / BLOCK_SIZE;
  int bb = BLOCK_SIZE;
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  //numBlocks = min((N - 1) / 16, (N + c0 - 2 )/ 16) - c0;
  for (int c0 = 0; c0 <= (N - 1)/bb; c0 += 1)  // serial loop
  {
    //for (int c1 = c0; c1 <= min((N - 1) / 16, (N + c0 - 2 )/ 16); c1 += 1) // parallel loop  blocks
    numBlocks = min((N - 1) / bb, (N + c0 - 2 )/ bb) - c0 + 1;
    myKernel<<<numBlocks, dimBlock>>>(d_S, n, c0, d_sequence);


    hipError_t errSync  = hipDeviceSynchronize();

    // Sprawdzenie błędów związanych z wywołaniem kernela (np. błędne parametry wywołania)
    hipError_t errAsync = hipGetLastError();

    // Sprawdzenie, czy pojawiły się błędy
    if (errSync != hipSuccess) {
      printf("Cuda synchronization error: %s\n", hipGetErrorString(errSync));
      exit(1);
    }

    if (errAsync != hipSuccess) {
      printf("Cuda asynchronous kernel error: %s\n", hipGetErrorString(errAsync));
      exit(1);
    }

  }

  hipMemcpy(&S[0][0], flat_d_S, n * n * sizeof(int), hipMemcpyDeviceToHost);

  double end_time = omp_get_wtime();
  double elapsed_time = end_time - start_time;
  printf("Time taken: %f seconds\n", elapsed_time);

  printf("gpu ended\n");


  cout << endl << endl;
  if(1==0)
  for(i=0; i<N; i++){
    for(j=0; j<N; j++){
      if(S[i][j] < 0)
        cout << "";
      else
        cout << S[i][j];
      cout << "\t";
    }
    cout << "\n";
  }
  cout << endl;


 // cpu control   loop uday dynamic tiling paper
  //if(1==0)
  for (i = N-1; i >= 0; i--) {
    for (j = i+1; j < N; j++) {
      for (k = 0; k < j-i; k++) {
        S_CPU[i][j] = max(S_CPU[i][k+i] + S_CPU[k+i+1][j], S_CPU[i][j]);
      }

      S_CPU[i][j] = max(S_CPU[i][j], S_CPU[i+1][j-1] + paired(seqq[i],seqq[j]));

    }
  }

  for(i=0; i<N; i++)
    for(j=0; j<N; j++)
      if(S[i][j] != S_CPU[i][j]){
        cout << i <<" " <<  j << ":" << S[i][j] << " " << S_CPU[i][j] << endl;
        cout << "error" << endl;
        exit(1);

      }


  delete[] S;
  delete[] S_CPU;

  hipFree(d_S);
  hipFree(flat_d_S);

  return 0;
}