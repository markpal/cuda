#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib> // for random numbers
#include <omp.h>
#include <vector>
#include <cstring> // for strcpy
#include <string>
#include <ctime>     // dla time()


#define BLOCK_SIZE 4
int N = 64;

using namespace std;


// -------------------------------------------------- pairing
int paired(char a1, char a2)
{
  if(a1 == 'A' && a2 == 'U')
    return 1;
  if(a1 == 'U' && a2 == 'A')
    return 1;
  if(a1 == 'G' && a2 == 'C')
    return 1;
  if(a1 == 'C' && a2 == 'G')
    return 1;

  return 0;
}

__device__ int _paired(char a, char b) {
  if ((a == 'A' && b == 'U') || (a == 'U' && b == 'A') || (a == 'C' && b == 'G') || (a == 'G' && b == 'C')) {
    return 1;
  }
  return 0;
}

// --------------------------------------------------
// KERNEL

__global__ void myKernel(int **B, int N, int c0, char* seqq)
{
        int c1 = blockIdx.x + c0;
        int bb = BLOCK_SIZE;
        __shared__ int C[BLOCK_SIZE][BLOCK_SIZE];
        C[threadIdx.y][threadIdx.x] = 0;

        if(c1 <= min((N - 1) / bb, (N + c0 - 2 )/ bb))
        //for (int c1 = c0; c1 <= min((N - 1) / 16, (N + c0 - 2 )/ 16); c1 += 1) // parallel loop  blocks
        {
            int _sj = c1-c0;
            int _si = c1;


           // printf("%i %i\n", _sj, _si);



         for (int m = _sj+1; m < _si; ++m) {

             if((threadIdx.x == 0) && (threadIdx.y == 0))
               {
              // printf("!!! BLOK %i %i | Poprzedni A %i %i oraz B %i %i !!! \n", _sj, _si, _sj, m, m, _si);

               }

               int row = threadIdx.y;
               int col = threadIdx.x;

              __shared__ int * A_elements[BLOCK_SIZE];
              __shared__ int * B_elements[BLOCK_SIZE];

           //   for(int i=0; i < BLOCK_SIZE; i++){
                A_elements[row] = &B[BLOCK_SIZE * _sj+row][BLOCK_SIZE * m -1];
                B_elements[row] = &B[BLOCK_SIZE * m +row][BLOCK_SIZE * _si];
             // }

              __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
              __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

             // Thread row and column


             if(row < BLOCK_SIZE && col < BLOCK_SIZE){

              int Cvalue = 0;
              As[row][col] =  A_elements[row][col];
              Bs[row][col] = B_elements[row][col];


              __syncthreads();

              for (int e = 0; e < BLOCK_SIZE; e++)
              {
                  Cvalue = max(As[row][e] + Bs[e][col], Cvalue);
              }

              __syncthreads();

                C[row][col] = max(C[row][col], Cvalue);

               __syncthreads();
               /*
               if((threadIdx.x == 0) && (threadIdx.y == 0) && (_si - _sj - 1 >= 1) && (_sj ==0) && (_si==3)){
                 printf("!!! BLOK %i %i | Poprzedni A %i %i oraz B %i %i !!! \n", _sj, _si, _sj, m, m, _si);

                 for(int y=0; y<BLOCK_SIZE; y++){
                   for(int x=0; x<BLOCK_SIZE; x++)
                     printf("%i ",C[y][x]);
                   printf("\n");
                 }}
               __syncthreads();*/
            }

           }
   /*       if((threadIdx.x == 0) && (threadIdx.y == 0) && (_si - _sj - 1 >= 1)){
            printf("!!! BLOK %i %i  !!! \n", _sj, _si);
         for(int y=0; y<BLOCK_SIZE; y++){
            for(int x=0; x<BLOCK_SIZE; x++)
              printf("%i ",C[y][x]);
            printf("\n");
          }} */
         // if(threadIdx.y ==0)
            for (int c2 = max(1, bb * c0 - bb - 1);
                 c2 <= min(bb * c0 + bb - 1, N + bb * c0 - bb * c1 - 1); c2 += 1) { // serial loop
                if (c0 >= 1) {
                    //    #pragma omp parallel for
                    int lb = max(bb * c1, -bb * c0 + bb * c1 + c2);
                    int ub = min(min(N - 1, bb * c1 + bb-1), -bb * c0 + bb * c1 + c2 + bb-1);
                    int c3 = threadIdx.x+ lb;
                    if(c3<=ub) {

                      register int z = B[-c2 + c3][c3];
                     // for (int c3 = max(16 * c1, -16 * c0 + 16 * c1 + c2); c3 <= min(min(N - 1, 16 * c1 + 15), -16 * c0 + 16 * c1 + c2 + 15); c3 += 1) {   // parallel loop threads

                      // !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1
                      if(1==1){

                  //   for (int c4 = bb-1; c4 < bound; c4 += 1) // serial
                            //  z = max(B[-c2 + c3][-c2 + c3 + c4  /* !!! */ - 1] + B[-c2 + c3 + c4 + 1 /* !!! */ - 1][c3], z);
                      // -----------------------------------------------------------------

                        //printf("%i %i %i %i\n", -c2+c3, c3, _j, _i);


                        //cout << _si << " | " << _sj << endl;

                          // przelicz bez y, tylko dla 1 watka ostatna kolumne i rzad
                        if(threadIdx.y ==0){

                          int _j = (-c2+c3) % BLOCK_SIZE;
                          int _i = c3 % BLOCK_SIZE;

                          if(_si - _sj - 1 >= 1)
                            z = max(z, C[_j][_i]);

                            int bound = ((c2) / bb) *bb -1;
                            int c4 = bound;

                            z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3], z);
                            // column block

                        for (int c4 = 0; c4 < bb-1; c4 += 1)  // blocks 0 (triangles)
                          z = max(B[-c2 + c3][-c2 + c3 + c4 ] + B[-c2 + c3 + c4 + 1][c3], z);

                        for (int c4 = bound+1; c4 < c2; c4 += 1)   // obecny blok
                          z = max(B[-c2 + c3][-c2 + c3 + c4 ] + B[-c2 + c3 + c4 + 1][c3], z);

                        B[-c2 + c3][c3] = max(z,
                                                B[-c2 + c3 + 1][c3 - 1] + _paired(seqq[-c2 + c3], seqq[c3]));
                          }
                      }

                      else // original generated code
                        {
                        for (int c4 = 0; c4 < c2; c4 += 1) {  // serial
                          z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3],  z);
                        }
                        B[-c2 + c3][c3] = max(z,
                                              B[-c2 + c3 + 1][c3 - 1] + _paired(seqq[-c2 + c3], seqq[c3]));
                        }
                      }

                } else {
                    //  #pragma omp parallel for
                   // printf("%i %i\n", _sj, _si);
                  int lb = bb * c1 + c2;
                  int ub = min(N - 1, bb * c1 + bb-1);
                  int c3 = threadIdx.x + lb;  // threadIdx.x
                  if(c3<=ub) {    // mozna dac ostra wtedy policzy  bez czwartej
                  //for (int c3 = 16 * c1 + c2; c3 <= min(N - 1, 16 * c1 + 15); c3 += 1) {   // parallel loop threads
                    register int z = B[-c2 + c3][c3];
                        for (int c4 = 0; c4 < c2; c4 += 1) {  // serial
                            z = max(B[-c2 + c3][-c2 + c3 + c4] + B[-c2 + c3 + c4 + 1][c3],  z);
                        }
                        B[-c2 + c3][c3] = max(z,
                                              B[-c2 + c3 + 1][c3 - 1] + _paired(seqq[-c2 + c3], seqq[c3]));
                        //if(c1==0)
                        //printf("%i %i %i\n", -c2+c3, c3, B[-c2 + c3][c3]);
                    }

                }
            }
        }


}




// --------------------------------------------------


int main() {

 // string seq = "UCGCUACCAUUGCUUCUAGACCUACGAAAUAGUCUCAUCUCUACGGCAGUAGUGCAUCUGUGUCGCGCUGUUCGUGAACCGAGACGUUGCAAGUCUUGUGUCAUUUAGGCGUAUGCACUGCUCUCCCU";
   string seq = "GUACGUACGUACGUACGUAC";
  //seq = "AGUCGAUCAGUCGUAUCGUACGCUAGC";
 // int N = seq.length();


  int n = N, i,j,k;

  char *seqq = new char[N+1];
  if(N>1) // no debug
   {
    char znaki[] = {'C', 'G', 'U', 'A'};
    srand(static_cast<unsigned int>(time(0)));

    for (int i = 0; i < N; i++) {
      seqq[i] = znaki[rand() % 4];  // Losowy wybór z zestawu 'C', 'G', 'U', 'A'
    }
   }
   cout << seqq << endl;
  std::strcpy(seqq, seq.c_str());          // Copy the string content   // use random data for given big N, comment this

  int* flatArray_S = new int[n * n];
  int* flatArray_S_CPU = new int[n * n];

  // Allocate 2D host array for CPU and GPU
  int** S = new int*[n];
  int** S_CPU = new int*[n];

  for(int i = 0; i < n; i++) {
    S[i] = &flatArray_S[i * n];
    S_CPU[i] = &flatArray_S_CPU[i * n];
  }
  // initialization
  for(i=0; i<N; i++) {
    for(j=0; j<N; j++){
      S[i][j] = INT_MIN;
      S_CPU[i][j] = INT_MIN;
    }
  }
  for(i=0; i<N; i++){
    S[i][i] = 0;
    S_CPU[i][i] = 0;
    if(i+1 < N) {
      S[i][i + 1] = 0;
      S[i+1][i] = 0;
      S_CPU[i][i+1] = 0;
      S_CPU[i+1][i] = 0;
    }
  }
  // -----------------------------

  // cuda memory allocation
  int* flat_d_S;
  int** d_S;
  char *d_sequence;

  double start_time = omp_get_wtime();
  hipMalloc(&d_sequence, n);
  hipMalloc(&flat_d_S, n * n * sizeof(int));
  hipMalloc(&d_S, n * sizeof(int*));

  int* h_S[n];  // copy flat_d_S pointers to vector on host and copy to d_S vector of pointers
  for(int i = 0; i < n; i++) {
    h_S[i] = flat_d_S + i * n;
  }
  hipMemcpy(d_S, h_S, n * sizeof(int*), hipMemcpyHostToDevice);
  hipMemcpy(d_sequence, seqq, n, hipMemcpyHostToDevice);
  // Copy host data to device before entering the loop
  hipMemcpy(flat_d_S, &S[0][0], n * n * sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = (n) / BLOCK_SIZE;
  int bb = BLOCK_SIZE;
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  //numBlocks = min((N - 1) / 16, (N + c0 - 2 )/ 16) - c0;
  for (int c0 = 0; c0 <= (N - 1)/bb; c0 += 1)  // serial loop
  {
    //for (int c1 = c0; c1 <= min((N - 1) / 16, (N + c0 - 2 )/ 16); c1 += 1) // parallel loop  blocks
    numBlocks = min((N - 1) / bb, (N + c0 - 2 )/ bb) - c0 + 1;
    myKernel<<<numBlocks, dimBlock>>>(d_S, n, c0, d_sequence);


    hipError_t errSync  = hipDeviceSynchronize();

    // Sprawdzenie błędów związanych z wywołaniem kernela (np. błędne parametry wywołania)
    hipError_t errAsync = hipGetLastError();

    // Sprawdzenie, czy pojawiły się błędy
    if (errSync != hipSuccess) {
      printf("Cuda synchronization error: %s\n", hipGetErrorString(errSync));
      exit(1);
    }

    if (errAsync != hipSuccess) {
      printf("Cuda asynchronous kernel error: %s\n", hipGetErrorString(errAsync));
      exit(1);
    }

  }

  hipMemcpy(&S[0][0], flat_d_S, n * n * sizeof(int), hipMemcpyDeviceToHost);

  double end_time = omp_get_wtime();
  double elapsed_time = end_time - start_time;
  printf("Time taken: %f seconds\n", elapsed_time);

  printf("gpu ended\n");


  cout << endl << endl;
  if(1==0)
  for(i=0; i<N; i++){
    for(j=0; j<N; j++){
      if(S[i][j] < 0)
        cout << "";
      else
        cout << S[i][j];
      cout << "\t";
    }
    cout << "\n";
  }
  cout << endl;


 // kontrola z cpu
  for (i = N-1; i >= 0; i--) {
    for (j = i+1; j < N; j++) {
      for (k = 0; k < j-i; k++) {
        S_CPU[i][j] = max(S_CPU[i][k+i] + S_CPU[k+i+1][j], S_CPU[i][j]);
      }

      S_CPU[i][j] = max(S_CPU[i][j], S_CPU[i+1][j-1] + paired(seqq[i],seqq[j]));

    }
  }


  for(i=0; i<N; i++)
    for(j=0; j<N; j++)
      if(S[i][j] != S_CPU[i][j]){
        cout << i <<" " <<  j << ":" << S[i][j] << " " << S_CPU[i][j] << endl;
        cout << "error" << endl;
        //exit(1);

      }


  delete[] S;
  delete[] S_CPU;

  hipFree(d_S);
  hipFree(flat_d_S);

  return 0;
}